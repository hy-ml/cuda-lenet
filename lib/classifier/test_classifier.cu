#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include "test_header.h"


#define GRID 512

float get_rand()
{
    return (float)(rand() % 10) / 10;
}

int main(void)
{
    float *data_h, *data_d, *out_h, *out_d, *out_h_from_d;
    int iN = 128;
    int oN = 64;
    int iNBytes = iN * sizeof(float), oNBytes = oN * sizeof(float);

    float *w_h, *w_d, *b_h, *b_d;

    // Memory allocate
    data_h = (float *)malloc(iNBytes);
    out_h = (float *)malloc(oNBytes);
    out_h_from_d = (float *)malloc(oNBytes);
    w_h = (float*)malloc(iN*oN*sizeof(float));
    b_h = (float*)malloc(oN*sizeof(float));

    // Initialize
    for (int i = 0; i < iN; i++) {
        data_h[i] = get_rand(); 
    }

    for (int i = 0; i < iN*oN; i++) {
        w_h[i] = get_rand();
    }


    for (int i = 0; i < oN; i++) {
        b_h[i] = get_rand();
    }

    // CUDA memory allocate
    hipMalloc((void **)&data_d, iNBytes);
    hipMalloc((void **)&out_d, oNBytes);
    hipMalloc((void **)&w_d, iN*oN*sizeof(float));
    hipMalloc((void **)&b_d, oN*sizeof(float));

    hipMemcpy(data_d, data_h, iNBytes, hipMemcpyHostToDevice);
    hipMemcpy(w_d, w_h, iN*oN*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(b_d, b_h, oN*sizeof(float), hipMemcpyHostToDevice);

    // Execute
    classifier(data_h, iN, data_h, oN, w_h, b_h);
    classifier <<< GRID, oN / GRID + 1 >>> (data_d, iN, data_d, oN, w_d, b_d, oN);

    hipMemcpy(out_h_from_d, out_d, oN * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < oN; i++) {
        printf("i: %d, CPU: %3.3f, GPU: %3.3f\n", i, out_h[i], out_h_from_d[i]);
        assert(abs(out_h[i] - out_h_from_d[i]) < 0.001);
    }
    printf("%d\n", oN);

    // Free 
    free(data_h); free(out_h); free(out_h_from_d); free(w_h); free(b_h);
    // CUDA free
    hipFree(data_d), hipFree(out_d), hipFree(w_d), hipFree(b_d);

    printf("Success Test\n");
    return 0;
}
